#include "hip/hip_runtime.h"
﻿/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include "help.hpp"
#include "utils.hpp"
#include "ArgParser.hpp"
#include "debug_utils.hpp"
#include "particle.h"
#include <stdlib.h>
#include <stdio.h>
#include "precomp.hpp"
#include <string>
#include <iomanip>
#include <iostream>
#include <sstream>
#include <chrono>

typedef union {
	uint8_t b[200];
	uint64_t q[25];
	uint32_t d[50];
} ethhash;

#define rotate64(x, s) ((x << s) | (x >> (64U - s)))
#define rotate32(x, s) ((x << s) | (x >> (32U - s)))
#define bswap32(n) (rotate32(n & 0x00FF00FFU, 24U)|(rotate32(n, 8U) & 0x00FF00FFU))



#define mul_hi(a, b) __umulhi(a, b)

// mod              = 0xfffffffffffffffffffffffffffffffffffffffffffffffffffffffefffffc2f
__device__ const mp_number mod              = { {0xfffffc2f, 0xfffffffe, 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff} };

// tripleNegativeGx = 0x92c4cc831269ccfaff1ed83e946adeeaf82c096e76958573f2287becbb17b196
__device__ const mp_number tripleNegativeGx = { {0xbb17b196, 0xf2287bec, 0x76958573, 0xf82c096e, 0x946adeea, 0xff1ed83e, 0x1269ccfa, 0x92c4cc83 } };

// doubleNegativeGy = 0x6f8a4b11b2b8773544b60807e3ddeeae05d0976eb2f557ccc7705edf09de52bf
//__device__ const mp_number doubleNegativeGy = { {0x09de52bf, 0xc7705edf, 0xb2f557cc, 0x05d0976e, 0xe3ddeeae, 0x44b60807, 0xb2b87735, 0x6f8a4b11} };

// negativeGy       = 0xb7c52588d95c3b9aa25b0403f1eef75702e84bb7597aabe663b82f6f04ef2777
__device__ const mp_number negativeGy       = { {0x04ef2777, 0x63b82f6f, 0x597aabe6, 0x02e84bb7, 0xf1eef757, 0xa25b0403, 0xd95c3b9a, 0xb7c52588 } };

// Multiprecision subtraction. Underflow signalled via return value.
__device__ mp_word mp_sub(mp_number * const r, const mp_number * const a, const mp_number * const b) {
	mp_word t, c = 0;

	for (mp_word i = 0; i < MP_WORDS; ++i) {
		t = a->d[i] - b->d[i] - c;
		c = t > a->d[i] ? 1 : (t == a->d[i] ? c : 0);

		r->d[i] = t;
	}

	return c;
}



// Multiprecision subtraction of the modulus saved in mod. Underflow signalled via return value.
__device__ mp_word mp_sub_mod(mp_number * const r) {
	mp_number mod = { {0xfffffc2fU, 0xfffffffeU, 0xffffffffU, 0xffffffffU, 0xffffffffU, 0xffffffffU, 0xffffffffU, 0xffffffffU} };

	mp_word t, c = 0;

	for (mp_word i = 0; i < MP_WORDS; ++i) {
		t = r->d[i] - mod.d[i] - c;
		c = t > r->d[i] ? 1 : (t == r->d[i] ? c : 0);

		r->d[i] = t;
	}

	return c;
}


__device__ void mp_mod_sub(mp_number* const r, const mp_number* const a, const mp_number* const b) {
	mp_word i, t, c = 0;

	for (i = 0; i < MP_WORDS; ++i) {
		t = a->d[i] - b->d[i] - c;
		c = t < a->d[i] ? 0 : (t == a->d[i] ? c : 1);

		r->d[i] = t;
	}

	if (c) {
		c = 0;
		for (i = 0; i < MP_WORDS; ++i) {
			r->d[i] += mod.d[i] + c;
			c = r->d[i] < mod.d[i] ? 1 : (r->d[i] == mod.d[i] ? c : 0);
		}
	}
}

__device__ void mp_mod_sub_const(mp_number* const r, const mp_number* const a, const mp_number* const b) {
	mp_word i, t, c = 0;

	for (i = 0; i < MP_WORDS; ++i) {
		t = a->d[i] - b->d[i] - c;
		c = t < a->d[i] ? 0 : (t == a->d[i] ? c : 1);

		r->d[i] = t;
	}

	if (c) {
		c = 0;
		for (i = 0; i < MP_WORDS; ++i) {
			r->d[i] += mod.d[i] + c;
			c = r->d[i] < mod.d[i] ? 1 : (r->d[i] == mod.d[i] ? c : 0);
		}
	}
}


__device__ void mp_mod_sub_gx(mp_number* const r, const mp_number* const a) {
	mp_word i, t, c = 0;

	t = a->d[0] - 0x16f81798U; c = t < a->d[0] ? 0 : (t == a->d[0] ? c : 1); r->d[0] = t;
	t = a->d[1] - 0x59f2815bU - c; c = t < a->d[1] ? 0 : (t == a->d[1] ? c : 1); r->d[1] = t;
	t = a->d[2] - 0x2dce28d9U - c; c = t < a->d[2] ? 0 : (t == a->d[2] ? c : 1); r->d[2] = t;
	t = a->d[3] - 0x029bfcdbU - c; c = t < a->d[3] ? 0 : (t == a->d[3] ? c : 1); r->d[3] = t;
	t = a->d[4] - 0xce870b07U - c; c = t < a->d[4] ? 0 : (t == a->d[4] ? c : 1); r->d[4] = t;
	t = a->d[5] - 0x55a06295U - c; c = t < a->d[5] ? 0 : (t == a->d[5] ? c : 1); r->d[5] = t;
	t = a->d[6] - 0xf9dcbbacU - c; c = t < a->d[6] ? 0 : (t == a->d[6] ? c : 1); r->d[6] = t;
	t = a->d[7] - 0x79be667eU - c; c = t < a->d[7] ? 0 : (t == a->d[7] ? c : 1); r->d[7] = t;

	if (c) {
		c = 0;
		for (i = 0; i < MP_WORDS; ++i) {
			r->d[i] += mod.d[i] + c;
			c = r->d[i] < mod.d[i] ? 1 : (r->d[i] == mod.d[i] ? c : 0);
		}
	}
}

// Multiprecision subtraction modulo M of G_y from a number.
// Specialization of mp_mod_sub in hope of performance gain.
__device__ void mp_mod_sub_gy(mp_number* const r, const mp_number* const a) {
	mp_word i, t, c = 0;

	t = a->d[0] - 0xfb10d4b8U; c = t < a->d[0] ? 0 : (t == a->d[0] ? c : 1); r->d[0] = t;
	t = a->d[1] - 0x9c47d08fU - c; c = t < a->d[1] ? 0 : (t == a->d[1] ? c : 1); r->d[1] = t;
	t = a->d[2] - 0xa6855419U - c; c = t < a->d[2] ? 0 : (t == a->d[2] ? c : 1); r->d[2] = t;
	t = a->d[3] - 0xfd17b448U - c; c = t < a->d[3] ? 0 : (t == a->d[3] ? c : 1); r->d[3] = t;
	t = a->d[4] - 0x0e1108a8U - c; c = t < a->d[4] ? 0 : (t == a->d[4] ? c : 1); r->d[4] = t;
	t = a->d[5] - 0x5da4fbfcU - c; c = t < a->d[5] ? 0 : (t == a->d[5] ? c : 1); r->d[5] = t;
	t = a->d[6] - 0x26a3c465U - c; c = t < a->d[6] ? 0 : (t == a->d[6] ? c : 1); r->d[6] = t;
	t = a->d[7] - 0x483ada77U - c; c = t < a->d[7] ? 0 : (t == a->d[7] ? c : 1); r->d[7] = t;

	if (c) {
		c = 0;
		for (i = 0; i < MP_WORDS; ++i) {
			r->d[i] += mod.d[i] + c;
			c = r->d[i] < mod.d[i] ? 1 : (r->d[i] == mod.d[i] ? c : 0);
		}
	}
}

// Multiprecision addition. Overflow signalled via return value.
__device__ mp_word mp_add(mp_number* const r, const mp_number* const a) {
	mp_word c = 0;

	for (mp_word i = 0; i < MP_WORDS; ++i) {
		r->d[i] += a->d[i] + c;
		c = r->d[i] < a->d[i] ? 1 : (r->d[i] == a->d[i] ? c : 0);
	}

	return c;
}

// Multiprecision addition of the modulus saved in mod. Overflow signalled via return value.
__device__ mp_word mp_add_mod(mp_number* const r) {
	mp_word c = 0;

	for (mp_word i = 0; i < MP_WORDS; ++i) {
		r->d[i] += mod.d[i] + c;
		c = r->d[i] < mod.d[i] ? 1 : (r->d[i] == mod.d[i] ? c : 0);
	}

	return c;
}

// Multiprecision addition of two numbers with one extra word each. Overflow signalled via return value.
__device__ mp_word mp_add_more(mp_number* const r, mp_word* const extraR, const mp_number* const a, const mp_word* const extraA) {
	const mp_word c = mp_add(r, a);
	*extraR += *extraA + c;
	return *extraR < *extraA ? 1 : (*extraR == *extraA ? c : 0);
}

// Multiprecision greater than or equal (>=) operator
__device__ mp_word mp_gte(const mp_number* const a, const mp_number* const b) {
	mp_word l = 0, g = 0;

	for (mp_word i = 0; i < MP_WORDS; ++i) {
		if (a->d[i] < b->d[i]) l |= (1 << i);
		if (a->d[i] > b->d[i]) g |= (1 << i);
	}

	return g >= l;
}

// Bit shifts a number with an extra word to the right one step
__device__ void mp_shr_extra(mp_number* const r, mp_word* const e) {
	r->d[0] = (r->d[1] << 31) | (r->d[0] >> 1);
	r->d[1] = (r->d[2] << 31) | (r->d[1] >> 1);
	r->d[2] = (r->d[3] << 31) | (r->d[2] >> 1);
	r->d[3] = (r->d[4] << 31) | (r->d[3] >> 1);
	r->d[4] = (r->d[5] << 31) | (r->d[4] >> 1);
	r->d[5] = (r->d[6] << 31) | (r->d[5] >> 1);
	r->d[6] = (r->d[7] << 31) | (r->d[6] >> 1);
	r->d[7] = (*e << 31) | (r->d[7] >> 1);
	*e >>= 1;
}

// Bit shifts a number to the right one step
__device__ void mp_shr(mp_number* const r) {
	r->d[0] = (r->d[1] << 31) | (r->d[0] >> 1);
	r->d[1] = (r->d[2] << 31) | (r->d[1] >> 1);
	r->d[2] = (r->d[3] << 31) | (r->d[2] >> 1);
	r->d[3] = (r->d[4] << 31) | (r->d[3] >> 1);
	r->d[4] = (r->d[5] << 31) | (r->d[4] >> 1);
	r->d[5] = (r->d[6] << 31) | (r->d[5] >> 1);
	r->d[6] = (r->d[7] << 31) | (r->d[6] >> 1);
	r->d[7] >>= 1;
}

// Multiplies a number with a word and adds it to an existing number with an extra word, overflow of the extra word is signalled in return value
// This is a special function only used for modular multiplication
__device__ mp_word mp_mul_word_add_extra(mp_number* const r, const mp_number* const a, const mp_word w, mp_word* const extra) {
	mp_word cM = 0; // Carry for multiplication
	mp_word cA = 0; // Carry for addition
	mp_word tM = 0; // Temporary storage for multiplication

	for (mp_word i = 0; i < MP_WORDS; ++i) {
		tM = (a->d[i] * w + cM);
		cM = mul_hi(a->d[i], w) + (tM < cM);

		r->d[i] += tM + cA;
		cA = r->d[i] < tM ? 1 : (r->d[i] == tM ? cA : 0);
	}

	*extra += cM + cA;
	return *extra < cM ? 1 : (*extra == cM ? cA : 0);
}

// Multiplies a number with a word, potentially adds modhigher to it, and then subtracts it from en existing number, no extra words, no overflow
// This is a special function only used for modular multiplication
__device__ void mp_mul_mod_word_sub(mp_number* const r, const mp_word w, const bool withModHigher) {
	// Having these numbers declared here instead of using the global values in __constant address space seems to lead
	// to better optimizations by the compiler on my GTX 1070.
	mp_number mod = { { 0xfffffc2f, 0xfffffffe, 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff} };
	mp_number modhigher = { {0x00000000, 0xfffffc2f, 0xfffffffe, 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff, 0xffffffff} };

	mp_word cM = 0; // Carry for multiplication
	mp_word cS = 0; // Carry for subtraction
	mp_word tS = 0; // Temporary storage for subtraction
	mp_word tM = 0; // Temporary storage for multiplication
	mp_word cA = 0; // Carry for addition of modhigher

	for (mp_word i = 0; i < MP_WORDS; ++i) {
		tM = (mod.d[i] * w + cM);
		cM = mul_hi(mod.d[i], w) + (tM < cM);

		tM += (withModHigher ? modhigher.d[i] : 0) + cA;
		cA = tM < (withModHigher ? modhigher.d[i] : 0) ? 1 : (tM == (withModHigher ? modhigher.d[i] : 0) ? cA : 0);

		tS = r->d[i] - tM - cS;
		cS = tS > r->d[i] ? 1 : (tS == r->d[i] ? cS : 0);

		r->d[i] = tS;
	}
}

// Modular multiplication. Based on Algorithm 3 (and a series of hunches) from this article:
// https://www.esat.kuleuven.be/cosic/publications/article-1191.pdf
// When I first implemented it I never encountered a situation where the additional end steps
// of adding or subtracting the modulo was necessary. Maybe it's not for the particular modulo
// used in secp256k1, maybe the overflow bit can be skipped in to avoid 8 subtractions and
// trade it for the final steps? Maybe the final steps are necessary but seldom needed?
// I have no idea, for the time being I'll leave it like this, also see the comments at the
// beginning of this document under the title "Cutting corners".
__device__ void mp_mod_mul(mp_number* const r, const mp_number* const X, const mp_number* const Y) {
	mp_number Z = { {0} };
	mp_word extraWord;

	for (int i = MP_WORDS - 1; i >= 0; --i) {
		// Z = Z * 2^32
		extraWord = Z.d[7]; Z.d[7] = Z.d[6]; Z.d[6] = Z.d[5]; Z.d[5] = Z.d[4]; Z.d[4] = Z.d[3]; Z.d[3] = Z.d[2]; Z.d[2] = Z.d[1]; Z.d[1] = Z.d[0]; Z.d[0] = 0;

		// Z = Z + X * Y_i
		bool overflow = mp_mul_word_add_extra(&Z, X, Y->d[i], &extraWord);

		// Z = Z - qM
		mp_mul_mod_word_sub(&Z, extraWord, overflow);
	}

	*r = Z;
}

// Modular inversion of a number.
__device__ void mp_mod_inverse(mp_number* const r) {
	mp_number A = { { 1 } };
	mp_number C = { { 0 } };
	mp_number v = mod;

	mp_word extraA = 0;
	mp_word extraC = 0;

	while (r->d[0] || r->d[1] || r->d[2] || r->d[3] || r->d[4] || r->d[5] || r->d[6] || r->d[7]) {
		while (!(r->d[0] & 1)) {
			mp_shr(r);
			if (A.d[0] & 1) {
				extraA += mp_add_mod(&A);
			}

			mp_shr_extra(&A, &extraA);
		}

		while (!(v.d[0] & 1)) {
			mp_shr(&v);
			if (C.d[0] & 1) {
				extraC += mp_add_mod(&C);
			}

			mp_shr_extra(&C, &extraC);
		}

		if (mp_gte(r, &v)) {
			mp_sub(r, r, &v);
			mp_add_more(&A, &extraA, &C, &extraC);
		}
		else {
			mp_sub(&v, &v, r);
			mp_add_more(&C, &extraC, &A, &extraA);
		}
	}

	while (extraC) {
		extraC -= mp_sub_mod(&C);
	}

	v = mod;
	mp_sub(r, &v, &C);
}


// Elliptical point addition
// Does not handle points sharing X coordinate, this is a deliberate design choice.
// For more information on this choice see the beginning of this file.
__device__ void point_add(point* const r, point* const p, point* const o) {
	mp_number tmp;
	mp_number newX;
	mp_number newY;

	mp_mod_sub(&tmp, &o->x, &p->x);

	mp_mod_inverse(&tmp);

	mp_mod_sub(&newX, &o->y, &p->y);
	mp_mod_mul(&tmp, &tmp, &newX);

	mp_mod_mul(&newX, &tmp, &tmp);
	mp_mod_sub(&newX, &newX, &p->x);
	mp_mod_sub(&newX, &newX, &o->x);

	mp_mod_sub(&newY, &p->x, &newX);
	mp_mod_mul(&newY, &newY, &tmp);
	mp_mod_sub(&newY, &newY, &p->y);

	r->x = newX;
	r->y = newY;
}




/* ------------------------------------------------------------------------ */
/* Profanity.                                                               */
/* ------------------------------------------------------------------------ */
typedef struct {
	uint32_t found;
	uint32_t foundId;
	uint8_t foundHash[20];
} result;

__device__ void profanity_init_seed(const point* const precomp, point* const p, bool* const pIsFirst, const size_t precompOffset, const uint64_t seed) {
	point o;

	for (uint8_t i = 0; i < 8; ++i) {
		const uint8_t shift = i * 8;
		const uint8_t byte = (seed >> shift) & 0xFF;

		if (byte) {
			o = precomp[precompOffset + i * 255 + byte - 1];
			if (*pIsFirst) {
				*p = o;
				*pIsFirst = false;
			}
			else {
				point_add(p, p, &o);
			}
		}
	}
}

#define PROFANITY_INVERSE_SIZE 255


__device__ void profanity_init(const point* const precomp, mp_number* const pDeltaX, mp_number* const pPrevLambda, result* const pResult, const uint64_t seed[4], const uint64_t seedX[4], const uint64_t seedY[4]) {
	const size_t id = (threadIdx.x + blockIdx.x * blockDim.x);

	/*
	point p = {
		.x = {.d = {
			seedX.x & 0xFFFFFFFF, seedX.x >> 32,
			seedX.y & 0xFFFFFFFF, seedX.y >> 32,
			seedX.z & 0xFFFFFFFF, seedX.z >> 32,
			seedX.w & 0xFFFFFFFF, seedX.w >> 32,
		}},
		.y = {.d = {
			seedY.x & 0xFFFFFFFF, seedY.x >> 32,
			seedY.y & 0xFFFFFFFF, seedY.y >> 32,
			seedY.z & 0xFFFFFFFF, seedY.z >> 32,
			seedY.w & 0xFFFFFFFF, seedY.w >> 32,
		}},
	};*/

	point p;
	p.x.d[0] = seedX[0] & 0xFFFFFFFF;
	p.x.d[1] = seedX[0] >> 32;
	p.x.d[2] = seedX[1] & 0xFFFFFFFF;
	p.x.d[3] = seedX[1] >> 32;
	p.x.d[4] = seedX[2] & 0xFFFFFFFF;
	p.x.d[5] = seedX[2] >> 32;
	p.x.d[6] = seedX[3] & 0xFFFFFFFF;
	p.x.d[7] = seedX[3] >> 32;
	p.y.d[0] = seedY[0] & 0xFFFFFFFF;
	p.y.d[1] = seedY[0] >> 32;
	p.y.d[2] = seedY[1] & 0xFFFFFFFF;
	p.y.d[3] = seedY[1] >> 32;
	p.y.d[4] = seedY[2] & 0xFFFFFFFF;
	p.y.d[5] = seedY[2] >> 32;
	p.y.d[6] = seedY[3] & 0xFFFFFFFF;
	p.y.d[7] = seedY[3] >> 32;

	point p_random;
	bool bIsFirst = true;

	mp_number tmp1, tmp2;
	point tmp3;

	// Calculate k*G where k = seed.wzyx (in other words, find the point indicated by the private key represented in seed)
	profanity_init_seed(precomp, &p_random, &bIsFirst, 8 * 255 * 0, seed[0]);
	profanity_init_seed(precomp, &p_random, &bIsFirst, 8 * 255 * 1, seed[1]);
	profanity_init_seed(precomp, &p_random, &bIsFirst, 8 * 255 * 2, seed[2]);
	profanity_init_seed(precomp, &p_random, &bIsFirst, 8 * 255 * 3, seed[3] + id);

	point_add(&p, &p, &p_random);

	// Calculate current lambda in this point
	mp_mod_sub_gx(&tmp1, &p.x);
	mp_mod_inverse(&tmp1);

	mp_mod_sub_gy(&tmp2, &p.y);
	mp_mod_mul(&tmp1, &tmp1, &tmp2);

	// Jump to next point (precomp[0] is the generator point G)
	tmp3 = precomp[0];
	point_add(&p, &tmp3, &p);

	// pDeltaX should contain the delta (x - G_x)
	mp_mod_sub_gx(&p.x, &p.x);

	pDeltaX[id] = p.x;
	pPrevLambda[id] = tmp1;

	for (uint8_t i = 0; i < 40 + 1; ++i) {
		pResult[i].found = 0;
	}
}


// This kernel calculates several modular inversions at once with just one inverse.
// It's an implementation of Algorithm 2.11 from Modern Computer Arithmetic:
// https://members.loria.fr/PZimmermann/mca/pub226.html
//
// My RX 480 is very sensitive to changes in the second loop and sometimes I have
// to make seemingly non-functional changes to the code to make the compiler
// generate the most optimized version.
__global__ void profanity_inverse(const mp_number* const pDeltaX, mp_number* const pInverse) {
	const size_t id = (threadIdx.x + blockIdx.x * blockDim.x) * PROFANITY_INVERSE_SIZE;

	// negativeDoubleGy = 0x6f8a4b11b2b8773544b60807e3ddeeae05d0976eb2f557ccc7705edf09de52bf
	mp_number negativeDoubleGy = { {0x09de52bf, 0xc7705edf, 0xb2f557cc, 0x05d0976e, 0xe3ddeeae, 0x44b60807, 0xb2b87735, 0x6f8a4b11 } };

	mp_number copy1, copy2;
	mp_number buffer[PROFANITY_INVERSE_SIZE];
	mp_number buffer2[PROFANITY_INVERSE_SIZE];

	// We initialize buffer and buffer2 such that:
	// buffer[i] = pDeltaX[id] * pDeltaX[id + 1] * pDeltaX[id + 2] * ... * pDeltaX[id + i]
	// buffer2[i] = pDeltaX[id + i]
	buffer[0] = pDeltaX[id];
	for (uint32_t i = 1; i < PROFANITY_INVERSE_SIZE; ++i) {
		buffer2[i] = pDeltaX[id + i];
		mp_mod_mul(&buffer[i], &buffer2[i], &buffer[i - 1]);
	}

	// Take the inverse of all x-values combined
	copy1 = buffer[PROFANITY_INVERSE_SIZE - 1];
	mp_mod_inverse(&copy1);

	// We multiply in -2G_y together with the inverse so that we have:
	//            - 2 * G_y
	//  ----------------------------
	//  x_0 * x_1 * x_2 * x_3 * ...
	mp_mod_mul(&copy1, &copy1, &negativeDoubleGy);

	// Multiply out each individual inverse using the buffers
	for (uint32_t i = PROFANITY_INVERSE_SIZE - 1; i > 0; --i) {
		mp_mod_mul(&copy2, &copy1, &buffer[i - 1]);
		mp_mod_mul(&copy1, &copy1, &buffer2[i]);
		pInverse[id + i] = copy2;
	}

	pInverse[id] = copy1;
}

__device__ void sha3_keccakf(ethhash* const h);

__global__ void profanity_iterate(mp_number* const pDeltaX, mp_number* const pInverse, mp_number* const pPrevLambda) {
	const size_t id = (threadIdx.x + blockIdx.x * blockDim.x);

	// negativeGx = 0x8641998106234453aa5f9d6a3178f4f8fd640324d231d726a60d7ea3e907e497
	mp_number negativeGx = { {0xe907e497, 0xa60d7ea3, 0xd231d726, 0xfd640324, 0x3178f4f8, 0xaa5f9d6a, 0x06234453, 0x86419981 } };

	ethhash h = { { 0 } };

	mp_number dX = pDeltaX[id];
	mp_number tmp = pInverse[id];
	mp_number lambda = pPrevLambda[id];

	// λ' = - (2G_y) / d' - λ <=> lambda := pInversedNegativeDoubleGy[id] - pPrevLambda[id]
	mp_mod_sub(&lambda, &tmp, &lambda);

	// λ² = λ * λ <=> tmp := lambda * lambda = λ²
	mp_mod_mul(&tmp, &lambda, &lambda);

	// d' = λ² - d - 3g = (-3g) - (d - λ²) <=> x := tripleNegativeGx - (x - tmp)
	mp_mod_sub(&dX, &dX, &tmp);
	mp_mod_sub_const(&dX, &tripleNegativeGx, &dX);

	pDeltaX[id] = dX;
	pPrevLambda[id] = lambda;

	// Calculate y from dX and lambda
	// y' = (-G_Y) - λ * d' <=> p.y := negativeGy - (p.y * p.x)
	mp_mod_mul(&tmp, &lambda, &dX);
	mp_mod_sub_const(&tmp, &negativeGy, &tmp);

	// Restore X coordinate from delta value
	mp_mod_sub(&dX, &dX, &negativeGx);

	// Initialize Keccak structure with point coordinates in big endian
	h.d[0] = bswap32(dX.d[MP_WORDS - 1]);
	h.d[1] = bswap32(dX.d[MP_WORDS - 2]);
	h.d[2] = bswap32(dX.d[MP_WORDS - 3]);
	h.d[3] = bswap32(dX.d[MP_WORDS - 4]);
	h.d[4] = bswap32(dX.d[MP_WORDS - 5]);
	h.d[5] = bswap32(dX.d[MP_WORDS - 6]);
	h.d[6] = bswap32(dX.d[MP_WORDS - 7]);
	h.d[7] = bswap32(dX.d[MP_WORDS - 8]);
	h.d[8] = bswap32(tmp.d[MP_WORDS - 1]);
	h.d[9] = bswap32(tmp.d[MP_WORDS - 2]);
	h.d[10] = bswap32(tmp.d[MP_WORDS - 3]);
	h.d[11] = bswap32(tmp.d[MP_WORDS - 4]);
	h.d[12] = bswap32(tmp.d[MP_WORDS - 5]);
	h.d[13] = bswap32(tmp.d[MP_WORDS - 6]);
	h.d[14] = bswap32(tmp.d[MP_WORDS - 7]);
	h.d[15] = bswap32(tmp.d[MP_WORDS - 8]);
	h.d[16] ^= 0x01; // length 64

	sha3_keccakf(&h);

	// Save public address hash in pInverse, only used as interim storage until next cycle
	pInverse[id].d[0] = h.d[3];
	pInverse[id].d[1] = h.d[4];
	pInverse[id].d[2] = h.d[5];
	pInverse[id].d[3] = h.d[6];
	pInverse[id].d[4] = h.d[7];
}


#define TH_ELT(t, c0, c1, c2, c3, c4, d0, d1, d2, d3, d4) \
{ \
    t = rotate64((uint64_t)(d0 ^ d1 ^ d2 ^ d3 ^ d4), (uint64_t)1) ^ (c0 ^ c1 ^ c2 ^ c3 ^ c4); \
}

#define THETA(s00, s01, s02, s03, s04, \
              s10, s11, s12, s13, s14, \
              s20, s21, s22, s23, s24, \
              s30, s31, s32, s33, s34, \
              s40, s41, s42, s43, s44) \
{ \
    TH_ELT(t0, s40, s41, s42, s43, s44, s10, s11, s12, s13, s14); \
    TH_ELT(t1, s00, s01, s02, s03, s04, s20, s21, s22, s23, s24); \
    TH_ELT(t2, s10, s11, s12, s13, s14, s30, s31, s32, s33, s34); \
    TH_ELT(t3, s20, s21, s22, s23, s24, s40, s41, s42, s43, s44); \
    TH_ELT(t4, s30, s31, s32, s33, s34, s00, s01, s02, s03, s04); \
    s00 ^= t0; s01 ^= t0; s02 ^= t0; s03 ^= t0; s04 ^= t0; \
    s10 ^= t1; s11 ^= t1; s12 ^= t1; s13 ^= t1; s14 ^= t1; \
    s20 ^= t2; s21 ^= t2; s22 ^= t2; s23 ^= t2; s24 ^= t2; \
    s30 ^= t3; s31 ^= t3; s32 ^= t3; s33 ^= t3; s34 ^= t3; \
    s40 ^= t4; s41 ^= t4; s42 ^= t4; s43 ^= t4; s44 ^= t4; \
}

#define RHOPI(s00, s01, s02, s03, s04, \
              s10, s11, s12, s13, s14, \
              s20, s21, s22, s23, s24, \
              s30, s31, s32, s33, s34, \
              s40, s41, s42, s43, s44) \
{ \
	t0  = rotate64(s10, (uint64_t) 1);  \
	s10 = rotate64(s11, (uint64_t)44); \
	s11 = rotate64(s41, (uint64_t)20); \
	s41 = rotate64(s24, (uint64_t)61); \
	s24 = rotate64(s42, (uint64_t)39); \
	s42 = rotate64(s04, (uint64_t)18); \
	s04 = rotate64(s20, (uint64_t)62); \
	s20 = rotate64(s22, (uint64_t)43); \
	s22 = rotate64(s32, (uint64_t)25); \
	s32 = rotate64(s43, (uint64_t) 8); \
	s43 = rotate64(s34, (uint64_t)56); \
	s34 = rotate64(s03, (uint64_t)41); \
	s03 = rotate64(s40, (uint64_t)27); \
	s40 = rotate64(s44, (uint64_t)14); \
	s44 = rotate64(s14, (uint64_t) 2); \
	s14 = rotate64(s31, (uint64_t)55); \
	s31 = rotate64(s13, (uint64_t)45); \
	s13 = rotate64(s01, (uint64_t)36); \
	s01 = rotate64(s30, (uint64_t)28); \
	s30 = rotate64(s33, (uint64_t)21); \
	s33 = rotate64(s23, (uint64_t)15); \
	s23 = rotate64(s12, (uint64_t)10); \
	s12 = rotate64(s21, (uint64_t) 6); \
	s21 = rotate64(s02, (uint64_t) 3); \
	s02 = t0; \
}

#define KHI(s00, s01, s02, s03, s04, \
            s10, s11, s12, s13, s14, \
            s20, s21, s22, s23, s24, \
            s30, s31, s32, s33, s34, \
            s40, s41, s42, s43, s44) \
{ \
    t0 = s00 ^ (~s10 &  s20); \
    t1 = s10 ^ (~s20 &  s30); \
    t2 = s20 ^ (~s30 &  s40); \
    t3 = s30 ^ (~s40 &  s00); \
    t4 = s40 ^ (~s00 &  s10); \
    s00 = t0; s10 = t1; s20 = t2; s30 = t3; s40 = t4; \
    \
    t0 = s01 ^ (~s11 &  s21); \
    t1 = s11 ^ (~s21 &  s31); \
    t2 = s21 ^ (~s31 &  s41); \
    t3 = s31 ^ (~s41 &  s01); \
    t4 = s41 ^ (~s01 &  s11); \
    s01 = t0; s11 = t1; s21 = t2; s31 = t3; s41 = t4; \
    \
    t0 = s02 ^ (~s12 &  s22); \
    t1 = s12 ^ (~s22 &  s32); \
    t2 = s22 ^ (~s32 &  s42); \
    t3 = s32 ^ (~s42 &  s02); \
    t4 = s42 ^ (~s02 &  s12); \
    s02 = t0; s12 = t1; s22 = t2; s32 = t3; s42 = t4; \
    \
    t0 = s03 ^ (~s13 &  s23); \
    t1 = s13 ^ (~s23 &  s33); \
    t2 = s23 ^ (~s33 &  s43); \
    t3 = s33 ^ (~s43 &  s03); \
    t4 = s43 ^ (~s03 &  s13); \
    s03 = t0; s13 = t1; s23 = t2; s33 = t3; s43 = t4; \
    \
    t0 = s04 ^ (~s14 &  s24); \
    t1 = s14 ^ (~s24 &  s34); \
    t2 = s24 ^ (~s34 &  s44); \
    t3 = s34 ^ (~s44 &  s04); \
    t4 = s44 ^ (~s04 &  s14); \
    s04 = t0; s14 = t1; s24 = t2; s34 = t3; s44 = t4; \
}

#define IOTA(s00, r) { s00 ^= r; }

__device__ uint64_t keccakf_rndc[24] = {
	0x0000000000000001, 0x0000000000008082, 0x800000000000808a,
	0x8000000080008000, 0x000000000000808b, 0x0000000080000001,
	0x8000000080008081, 0x8000000000008009, 0x000000000000008a,
	0x0000000000000088, 0x0000000080008009, 0x000000008000000a,
	0x000000008000808b, 0x800000000000008b, 0x8000000000008089,
	0x8000000000008003, 0x8000000000008002, 0x8000000000000080,
	0x000000000000800a, 0x800000008000000a, 0x8000000080008081,
	0x8000000000008080, 0x0000000080000001, 0x8000000080008008
};

// Barely a bottleneck. No need to tinker more.
__device__ void sha3_keccakf(ethhash* const h)
{
//04
//4b90f6d1ab41f09f63f99a67fdcdd5494534c3a3f7d3a012883acc4af5f674b
//bd9dfa4e853140c516908a62666e292312b0108b5fde5754320e5d57971cc8d
//80
    uint64_t * const st = (uint64_t *) h;
	h->d[33] ^= 0x80000000;
	uint64_t t0, t1, t2, t3, t4;

	// Unrolling and removing PI stage gave negligable performance on GTX 1070.
	for (int i = 0; i < 24; ++i) {
		THETA(st[0], st[5], st[10], st[15], st[20], st[1], st[6], st[11], st[16], st[21], st[2], st[7], st[12], st[17], st[22], st[3], st[8], st[13], st[18], st[23], st[4], st[9], st[14], st[19], st[24]);
		RHOPI(st[0], st[5], st[10], st[15], st[20], st[1], st[6], st[11], st[16], st[21], st[2], st[7], st[12], st[17], st[22], st[3], st[8], st[13], st[18], st[23], st[4], st[9], st[14], st[19], st[24]);
		KHI(st[0], st[5], st[10], st[15], st[20], st[1], st[6], st[11], st[16], st[21], st[2], st[7], st[12], st[17], st[22], st[3], st[8], st[13], st[18], st[23], st[4], st[9], st[14], st[19], st[24]);
		IOTA(st[0], keccakf_rndc[i]);
	}
}

__global__ void sha3_keccakf_host(ethhash* const ethash_data)
{
	const size_t id = (threadIdx.x + blockIdx.x * blockDim.x);

    sha3_keccakf(&ethash_data[id]);
}

__global__ void sha3_keccakf_host2(int32_t* const ethash_data)
{
	const size_t id = (threadIdx.x + blockIdx.x * blockDim.x);
    ethhash h = { 0 };

//df5f587d7a24f070e0d289a31f9d40d3856173ef0f163b8a3d735f7e72bab46172b4f18429d1fc3d55d86b1d9adc56f0d2e8f85d664ca6ff2faa195d686f8dd6

    h.d[0] = 0x7d585fdf;
    h.d[1] = 0x70f0247a;
    h.d[2] = 0x9da3890d;
    h.d[3] = 0x85d4409f;
    h.d[4] = 0xef737685;
    h.d[5] = 0x3b160fef;
    h.d[6] = 0x1f9a289d;
    h.d[7] = 0x61d4ba72;
    h.d[8] = 0x4f1842b4;
    h.d[9] = 0x3d1fcd9d;
    h.d[10] = 0x7e5f733d;
    h.d[11] = 0x61b4ba72;
    h.d[12] = 0x6fca64d6;
    h.d[13] = 0x2f6fca6f;
    h.d[14] = id / 0xffffffff;
    h.d[15] = id % 0xffffffff;
    h.d[16] = 0x00000001;
    sha3_keccakf(&h);

    ethash_data[id] = h.d[0] ^ h.d[1] ^ h.d[2] ^ h.d[3] ^ h.d[4] ^ h.d[5] ^ h.d[6] ^ h.d[7] ^ h.d[8];
}

__global__ void advanceParticlesPart1(float dt, particle * pArray, point* precomp, mp_number* pointsDeltaX, mp_number* pPrevLambda, mp_number* pInverse,
    uint64_t seedX1, uint64_t seedX2, uint64_t seedX3, uint64_t seedX4, uint64_t seedY1, uint64_t seedY2, uint64_t seedY3, uint64_t seedY4)
{
	uint64_t seed[4];
	seed[0] = 3;
	seed[1] = 1;
	seed[2] = 1;
	seed[3] = 1;
	uint64_t seedX[4];
	seedX[0] = seedX1;
	seedX[1] = seedX2;
	seedX[2] = seedX3;
	seedX[3] = seedX4;
	uint64_t seedY[4];
	seedY[0] = seedY1;
	seedY[1] = seedY2;
	seedY[2] = seedY3;
	seedY[3] = seedY4;
	result pResult = { 0 };



	profanity_init(precomp, pointsDeltaX, pPrevLambda, &pResult, seed, seedX, seedY);


	//pInverse[(threadIdx.x + blockIdx.x * blockDim.x) * PROFANITY_INVERSE_SIZE].d[0] = (uint32_t)(seedX1 & 0x00000000FFFFFFFFU);
}


static std::string toHex(const uint8_t* const s, const size_t len) {
	std::string b("0123456789abcdef");
	std::string r;

	for (size_t i = 0; i < len; ++i) {
		const unsigned char h = s[i] / 16;
		const unsigned char l = s[i] % 16;

		r = r + b.substr(h, 1) + b.substr(l, 1);
	}

	return r;
}

static void printResult(const uint64_t seed[4], uint64_t round, result r, uint8_t score) {
	// Format private key
	uint64_t carry = 0;
	uint64_t seedRes[4];

	seedRes[0] = seed[0] + round;
	carry = seedRes[0] < round;
	seedRes[1] = seed[1] + carry;
	carry = !seedRes[1];
	seedRes[2] = seed[2] + carry;
	carry = !seedRes[2];
	seedRes[3] = seed[3] + carry + r.foundId;

	std::ostringstream ss;
	ss << std::hex << std::setfill('0');
	ss << std::setw(16) << seedRes[3] << std::setw(16) << seedRes[2] << std::setw(16) << seedRes[1] << std::setw(16) << seedRes[0];
	const std::string strPrivate = ss.str();

	// Format public key
	const std::string strPublic = toHex(r.foundHash, 20);

	// Print
	std::cout << "s Score: " << std::setw(2) << (int)score << " Private: 0x" << strPrivate << ' ';

	std::cout << ": 0x" << strPublic << std::endl;
}

struct PublicKeyPart {
    uint64_t val[4];
};

PublicKeyPart hexStringToUint64(const std::string& hexStr) {
    // Ensure the input string is not too long
    if (hexStr.length() != 64) {
        throw std::invalid_argument("Hex string has to be exactly 64 characters.");
    }
    // Function to convert a 16-character substring to a uint64_t
    auto hexToUint64 = [](const std::string& subHex) -> uint64_t {
        uint64_t value = 0;
        std::istringstream iss(subHex);
        iss >> std::hex >> value;
        return value;
    };

    PublicKeyPart result;
    // Extract 16-character chunks and convert them to uint64_t
    result.val[0] = hexToUint64(hexStr.substr(0, 16));
    result.val[1] = hexToUint64(hexStr.substr(16, 16));
    result.val[2] = hexToUint64(hexStr.substr(32, 16));
    result.val[3] = hexToUint64(hexStr.substr(48, 16));
    return result;
}

int main(int argc, char ** argv)
{

		ArgParser argp(argc, argv);
		bool bHelp = false;
		bool bModeBenchmark = false;
		bool bModeZeros = false;
		bool bModeZeroBytes = false;
		bool bModeLetters = false;
		bool bModeNumbers = false;
		std::string strModeLeading;
		std::string strModeMatching;
		std::string strPublicKey;
		bool bModeLeadingRange = false;
		bool bModeRange = false;
		bool bModeMirror = false;
		bool bModeDoubles = false;
		int rangeMin = 0;
		int rangeMax = 0;
		std::vector<size_t> vDeviceSkipIndex;
		size_t worksizeLocal = 64;
		size_t worksizeMax = 0; // Will be automatically determined later if not overriden by user
		bool bNoCache = false;
		size_t inverseSize = 255;
		size_t inverseMultiple = 16384;
		bool bMineContract = false;

		argp.addSwitch('h', "help", bHelp);
		argp.addSwitch('0', "benchmark", bModeBenchmark);
		argp.addSwitch('1', "zeros", bModeZeros);
		argp.addSwitch('2', "letters", bModeLetters);
		argp.addSwitch('3', "numbers", bModeNumbers);
		argp.addSwitch('4', "leading", strModeLeading);
		argp.addSwitch('5', "matching", strModeMatching);
		argp.addSwitch('6', "leading-range", bModeLeadingRange);
		argp.addSwitch('7', "range", bModeRange);
		argp.addSwitch('8', "mirror", bModeMirror);
		argp.addSwitch('9', "leading-doubles", bModeDoubles);
		argp.addSwitch('m', "min", rangeMin);
		argp.addSwitch('M', "max", rangeMax);
		argp.addMultiSwitch('s', "skip", vDeviceSkipIndex);
		argp.addSwitch('w', "work", worksizeLocal);
		argp.addSwitch('W', "work-max", worksizeMax);
		argp.addSwitch('n', "no-cache", bNoCache);
		argp.addSwitch('i', "inverse-size", inverseSize);
		argp.addSwitch('I', "inverse-multiple", inverseMultiple);
		argp.addSwitch('c', "contract", bMineContract);
		argp.addSwitch('z', "publicKey", strPublicKey);
		argp.addSwitch('b', "zero-bytes", bModeZeroBytes);

		if (!argp.parse()) {
			std::cout << "error: bad arguments, -h for help" << std::endl;
			return 1;
		}
        if (bHelp) {
            std::cout << g_strHelp << std::endl;
            return 0;
        }
		if (strPublicKey.length() == 0) {
			std::cout << "error: this tool requires your public key to derive it's private key security" << std::endl;
			return 1;
		}
        strPublicKey = string_replace(strPublicKey, "0x", "");
        if (strPublicKey.length() != 128) {
            std::cout << "error: public key must be 128 hexadecimal characters long" << std::endl;
            return 1;
        }


        PublicKeyPart publicKeyX = hexStringToUint64(strPublicKey.substr(0, 64));
        PublicKeyPart publicKeyY = hexStringToUint64(strPublicKey.substr(64, 64));



	hipError_t error;
	const int run_size = 256;

	error = hipGetLastError();
	if (error != hipSuccess)
  	{
  	    printf("0 %s\n",hipGetErrorString(error));
  	    exit(1);
  	}

	particle * pArray = new particle[run_size];
	particle* devPArray = NULL;
	point * precomp = NULL;
	mp_number* pointsDeltaX = NULL;
	mp_number* prevLambda = NULL;
	mp_number* invData = NULL;
	hipMalloc(&devPArray, run_size*sizeof(particle));
	hipMalloc(&precomp, 8160 * sizeof(point));
	hipMalloc(&pointsDeltaX, PROFANITY_INVERSE_SIZE * run_size * sizeof(mp_number));
	hipMalloc(&prevLambda, PROFANITY_INVERSE_SIZE * run_size * sizeof(mp_number));
	hipMalloc(&invData, PROFANITY_INVERSE_SIZE * run_size * sizeof(mp_number));

    printf("Generating test data...\n");

    uint8_t public_key[64];
    const char* test_public_key = "65b3b3a2d97271fee54c747f796f123e5895a4bc096016fbc5163c8f51084ae8e8cda24b16cc02f0f8a33e8d890d7212d113d2ee33202d416f6401cc7614e85d";
    for (int i = 0; i < 64; i++) {
        std::stringstream ss;
        ss << std::hex << std::string(test_public_key + i * 2, 2);
        int byte;
        ss >> byte;
        public_key[i] = static_cast<uint8_t>(byte);
    }

    const int ethash_count = 25600000;
    ethhash* h = new ethhash[ethash_count]();
    for (int n = 0; n < ethash_count; n++) {
        for (int i = 0; i < 25; i++) {
            h[n].q[i] = 0;
        }
        memcpy(h[n].b, public_key, 64);
        h[n].b[64] = 0x01;
    }


    ethhash* deviceHash = NULL;
    hipMalloc(&deviceHash, sizeof(ethhash) * ethash_count);
    printf("Copying data to device...\n");

    hipMemcpy(deviceHash, h, sizeof(ethhash) * ethash_count, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    error = hipGetLastError();
	if (error != hipSuccess)
  	{
        printf("Initialize keccak test failed %s\n",hipGetErrorString(error));
        exit(1);
  	}
    printf("Running keccak...\n");
    auto start = std::chrono::high_resolution_clock::now();
    const uint64_t current_time = time(NULL);
  	sha3_keccakf_host2<<<ethash_count / 64, 64>>>((int*)deviceHash);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);

    // Output the duration
    std::cout << "Time taken: " << duration.count() / 1000.0 / 1000.0 << " ms" << std::endl;

    printf("Copying data back...\n");
    hipMemcpy(h, deviceHash, ethash_count * sizeof(ethhash), hipMemcpyDeviceToHost);
    error = hipGetLastError();
    if (error != hipSuccess)
  	{
        printf("Initialize keccak test failed %s\n",hipGetErrorString(error));
        exit(1);
  	}

    printf("Public key: ");
    for (int i = 12; i < 32; i++) {
        printf("%02x", h[ethash_count - 1].b[i]);
    }
    printf("\n");

    return 1;



	hipDeviceSynchronize(); error = hipGetLastError();
	if (error != hipSuccess)
  	{
        printf("1 %s\n",hipGetErrorString(error));
        exit(1);
  	}

	mp_number* pointsDeltaXHost = new mp_number[PROFANITY_INVERSE_SIZE * run_size];
	for(int i=0; i< PROFANITY_INVERSE_SIZE * run_size; i++)
	{
		for(int j=0; j<8; j++)
		{
			pointsDeltaXHost[i].d[j] = 0;
		}
	}

	mp_number* prevLambdaHost = new mp_number[PROFANITY_INVERSE_SIZE * run_size];
	for(int i=0; i< PROFANITY_INVERSE_SIZE * run_size; i++)
	{
		for(int j=0; j<8; j++)
		{
			prevLambdaHost[i].d[j] = 0;
		}
	}

	mp_number* invDataHost = new mp_number[PROFANITY_INVERSE_SIZE * run_size];
	for(int i=0; i< PROFANITY_INVERSE_SIZE * run_size; i++)
	{
		for(int j=0; j<8; j++)
		{
			invDataHost[i].d[j] = 0;
		}
	}

	hipMemcpy(devPArray, pArray, run_size*sizeof(particle), hipMemcpyHostToDevice);
	hipMemcpy(precomp, g_precomp, 8160 * sizeof(point), hipMemcpyHostToDevice);
	hipMemcpy(pointsDeltaX, pointsDeltaXHost, PROFANITY_INVERSE_SIZE * run_size*sizeof(mp_number), hipMemcpyHostToDevice);
	hipMemcpy(prevLambda, prevLambdaHost, PROFANITY_INVERSE_SIZE * run_size*sizeof(mp_number), hipMemcpyHostToDevice);
	hipMemcpy(invData, invDataHost, PROFANITY_INVERSE_SIZE * run_size*sizeof(mp_number), hipMemcpyHostToDevice);

	hipDeviceSynchronize(); error = hipGetLastError();
	if (error != hipSuccess)
  	{
        printf("2 %s\n",hipGetErrorString(error));
        exit(1);
  	}

	float dt = (float)rand()/(float) RAND_MAX; // Random distance each step
	advanceParticlesPart1<<< 1, 256>>>(dt, devPArray, precomp, pointsDeltaX, prevLambda, invData,
	publicKeyX.val[0],
	publicKeyX.val[1],
    	publicKeyX.val[2],
        	publicKeyX.val[3],
            	publicKeyY.val[1],
            	publicKeyY.val[2],
            	publicKeyY.val[3],
            	publicKeyY.val[4]
	);
	hipDeviceSynchronize();

	hipMemcpy(pArray, devPArray, run_size*sizeof(particle), hipMemcpyDeviceToHost);
	hipMemcpy(pointsDeltaXHost, pointsDeltaX, PROFANITY_INVERSE_SIZE * run_size*sizeof(mp_number), hipMemcpyDeviceToHost);
	hipMemcpy(prevLambdaHost, prevLambda, PROFANITY_INVERSE_SIZE * run_size*sizeof(mp_number), hipMemcpyDeviceToHost);
	hipMemcpy(invDataHost, invData, PROFANITY_INVERSE_SIZE * run_size*sizeof(mp_number), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("Delta host no: %d\n", i);
        pretty_print_mp_number(pointsDeltaXHost[i]);
        printf("\n");
    }
    for (int i = 0; i < 10; i++) {
        printf("Prev prevLambdaHost: %d\n", i);
        pretty_print_mp_number(prevLambdaHost[i]);
        printf("\n");
    }
    for (int i = 0; i < 10; i++) {
        printf("Inv data no: %d\n", i);
        pretty_print_mp_number(invDataHost[i]);
        printf("\n");
    }

	hipDeviceSynchronize();
	profanity_inverse<<< 1, 1>>>(pointsDeltaX, invData);



    error = hipGetLastError();
	hipMemcpy(pArray, devPArray, run_size*sizeof(particle), hipMemcpyDeviceToHost);
	hipMemcpy(pointsDeltaXHost, pointsDeltaX, PROFANITY_INVERSE_SIZE * run_size*sizeof(mp_number), hipMemcpyDeviceToHost);
	hipMemcpy(prevLambdaHost, prevLambda, PROFANITY_INVERSE_SIZE * run_size*sizeof(mp_number), hipMemcpyDeviceToHost);
	hipMemcpy(invDataHost, invData, PROFANITY_INVERSE_SIZE * run_size*sizeof(mp_number), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("Delta host no: %d\n", i);
        pretty_print_mp_number(pointsDeltaXHost[i]);
        printf("\n");
    }
    for (int i = 0; i < 10; i++) {
        printf("Prev prevLambdaHost: %d\n", i);
        pretty_print_mp_number(prevLambdaHost[i]);
        printf("\n");
    }
    for (int i = 0; i < 10; i++) {
        printf("Inv data no: %d\n", i);
        pretty_print_mp_number(invDataHost[i]);
        printf("\n");
    }

    if (error != hipSuccess)
    {
        printf("profanity_inverse error: %s\n",hipGetErrorString(error));
        exit(1);
    }
    hipDeviceSynchronize();

	profanity_iterate<<< 1, 256>>>(pointsDeltaX, invData, prevLambda);
	hipDeviceSynchronize();
	profanity_inverse <<<  1, 1 >>>  (pointsDeltaX, invData);
	hipDeviceSynchronize();
	profanity_iterate <<<  1, 256 >>>  (pointsDeltaX, invData, prevLambda);

	error = hipGetLastError();
	if (error != hipSuccess)
    {
        printf("profanity_iterate error %s\n",hipGetErrorString(error));
        exit(1);
    }
    	hipMemcpy(pArray, devPArray, run_size*sizeof(particle), hipMemcpyDeviceToHost);
    	hipMemcpy(pointsDeltaXHost, pointsDeltaX, PROFANITY_INVERSE_SIZE * run_size*sizeof(mp_number), hipMemcpyDeviceToHost);
    	hipMemcpy(prevLambdaHost, prevLambda, PROFANITY_INVERSE_SIZE * run_size*sizeof(mp_number), hipMemcpyDeviceToHost);
    	hipMemcpy(invDataHost, invData, PROFANITY_INVERSE_SIZE * run_size*sizeof(mp_number), hipMemcpyDeviceToHost);

        for (int i = 0; i < 10; i++) {
            printf("Delta host no: %d\n", i);
            pretty_print_mp_number(pointsDeltaXHost[i]);
            printf("\n");
        }
        for (int i = 0; i < 10; i++) {
            printf("Prev prevLambdaHost: %d\n", i);
            pretty_print_mp_number(prevLambdaHost[i]);
            printf("\n");
        }
        for (int i = 0; i < 10; i++) {
            printf("Inv data no: %d\n", i);
            pretty_print_mp_number(invDataHost[i]);
            printf("\n");
        }

    //return 1;

	hipDeviceSynchronize();

	printf("Size of mp_number %lld\n", sizeof(mp_number));
	hipMemcpy(pArray, devPArray, run_size*sizeof(particle), hipMemcpyDeviceToHost);
	hipMemcpy(pointsDeltaXHost, pointsDeltaX, PROFANITY_INVERSE_SIZE * run_size*sizeof(mp_number), hipMemcpyDeviceToHost);
	hipMemcpy(prevLambdaHost, prevLambda, PROFANITY_INVERSE_SIZE * run_size*sizeof(mp_number), hipMemcpyDeviceToHost);
	hipMemcpy(invDataHost, invData, PROFANITY_INVERSE_SIZE * run_size*sizeof(mp_number), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess)
    {
        printf("4 %s\n",hipGetErrorString(error));
        exit(1);
    }



	/*for (int n = 0; n < 100; n++)
	{
		printf("Hash no: %d\n", n);
		for (int i = 0; i < 32; i++)
		{
			printf("%d ", pArray[n].m_data[i]);
		}
		printf("\n");
	}*/
	for (uint64_t n = 0; n < 10; n++)
	{
		printf("Hash no: %lld\n0x", n);
		const uint8_t* hash = (uint8_t * )invDataHost[n].d;
		const uint64_t seed[4] = {1, 1, 1, 1 + n};
		result r = {0};
		r.found = 1;
		r.foundId = (uint32_t) n;
		memcpy(r.foundHash, hash, 20);
		printResult(seed, 2, r, 0);
		for (int i = 0; i < 20; i++)
		{
			printf("%02x", hash[i]);
		}
		printf("\n");
	}





	return 0;
}
